#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <math.h>

void initializeMatrix(float* Array, int SIZE) {
    // Function associates a random float at each index of the matrix
    for (int i = 0; i < SIZE; i++) {
        for (int j = 0; j < SIZE; j++) {
            Array[i * SIZE + j] = (float)(rand() / RAND_MAX);

        }
    }
}

void transferFunction(int SIZE) {
    // Initialize byte size by matrix size
    int BYTES = SIZE * SIZE * sizeof(float);
       
    // Initialize and allocate memory to host Matrix A and B
    float* hostMatrixA = 0;
    float* hostMatrixB = 0;
    hipHostMalloc((void**)&hostMatrixA, BYTES);
    hipHostMalloc((void**)&hostMatrixB, BYTES);
    initializeMatrix(hostMatrixA, SIZE);
    initializeMatrix(hostMatrixB, SIZE);

    // Initialize and allocate memory to device Matrix A and B
    float* deviceMatrixA = 0;
    float* deviceMatrixB = 0;
    hipMalloc((void**)&deviceMatrixA, BYTES);
    hipMalloc((void**)&deviceMatrixB, BYTES);

    // Initialize host to device and device to host times
    float HTD = 0;
    float DTH = 0;
    
    // CUDA event variables
    hipEvent_t startTime, stopTime;
    hipEventCreate(&startTime);
    hipEventCreate(&stopTime);
    hipDeviceSynchronize();

    // Start recording and copy the host matrices from the host to the device
    hipEventRecord(startTime, 0);
    hipMemcpy(deviceMatrixA, hostMatrixA, BYTES, hipMemcpyHostToDevice);
    hipMemcpy(deviceMatrixB, hostMatrixB, BYTES, hipMemcpyHostToDevice);

    // Stop the timer and associate it with host to device variable
    hipEventRecord(stopTime);
    hipEventSynchronize(stopTime);
    hipEventElapsedTime(&HTD, startTime, stopTime);

    // Start recording and copy the device matrices from device to host
    hipEventRecord(startTime, 0);
    hipMemcpy(hostMatrixA, deviceMatrixA, BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(hostMatrixB, deviceMatrixB, BYTES, hipMemcpyDeviceToHost);

    // Stop the timer and associate it with device to host variable
    hipEventRecord(stopTime); 
    hipEventSynchronize(stopTime);
    hipEventElapsedTime(&DTH, startTime, stopTime);

    printf("Host to Device for %d matrix size: % .2fms\n", SIZE, HTD);
    printf("Device to Host for %d matrix size: %.2fms\n", SIZE, DTH);

    // Free allocated memory and reset time events 
    hipEventDestroy(startTime);
    hipEventDestroy(stopTime);
    hipHostFree(hostMatrixA);
    hipHostFree(hostMatrixB);
    hipFree(deviceMatrixA);
    hipFree(deviceMatrixB);
    hipDeviceReset();
}

void matrixMul(float* resultMatrix, float* MatrixA, float* MatrixB, int SIZE) {
    
    // Initialize clock 
    clock_t startTime, stopTime;
    startTime = clock();

    // Matrix multiplication algorithm
    for (int i = 0; i < SIZE; i++) {
        for (int j = 0; j < SIZE; j++) {
            resultMatrix[i * SIZE + j] = 0;

            for (int k = 0; k < SIZE; k++) {
                resultMatrix[i * SIZE + j] += MatrixA[i * SIZE + k] * MatrixB[k * SIZE + j];
            }
        }
    }

    // Stop the time and print results
    stopTime = clock();
    printf("CPU Matrix Multiplication % .2fms\n", (double)stopTime - startTime);

}
__global__ void kernelMatrixMul(float* Result, float* MatrixA, float* MatrixB, int SIZE) {

    for (int i = 0; i < SIZE; i++) {
        for (int j = 0; j < SIZE; j++) {
            float matrixSum = 0;
            for (int k = 0; k < SIZE; k++) {
                matrixSum += MatrixA[i * SIZE + k] * MatrixB[k * SIZE + j];
            }
            Result[i * SIZE + j] = matrixSum;
        }
    }


    return;
}


void verifyMatrix(float* CPUMatrix, float* GPUMatrix, int SIZE) {

    // For every matrix index, check if the CPU and GPU results match within an allowance of 0.01
    for (int i = 0; i < SIZE; i++) {
        for (int j = 0; j < SIZE; j++) {
            if (fabs(CPUMatrix[i * SIZE + j] - GPUMatrix[i * SIZE + j]) > 0.01) {
                printf("TEST FAILED\n");
                return;
            }
        }
    }
    // Print passed if matrices match
    printf("TEST PASSED\n");
    return;

}

void gpuMatrixMul(float* Result, float* MatrixA, float* MatrixB, int SIZE) {
    // Determine byte size of matrix 
    int BYTES = SIZE * SIZE * sizeof(float);

    // Define device matrices and allocate them memory
    float* deviceMatrixA;
    float* deviceMatrixB;
    float* deviceResultMatrix;

    hipMalloc(&deviceMatrixA, BYTES);
    hipMalloc(&deviceMatrixB, BYTES);
    hipMalloc(&deviceResultMatrix, BYTES);

    // Copy the matrices from the host to the device
    hipMemcpy(deviceMatrixA, MatrixA, BYTES, hipMemcpyHostToDevice);
    hipMemcpy(deviceMatrixB, MatrixB, BYTES, hipMemcpyHostToDevice);

    // Establish time and CUDA events
    float time = 0;

    hipEvent_t startTime, stopTime;
    hipEventCreate(&startTime);
    hipEventCreate(&stopTime);
    hipDeviceSynchronize();

    // Make grid and block dimensions 1 
    dim3 threadsPerBlock(1, 1);
    dim3 blocksPerGrid(1, 1);

    // Start recording and start matrix multiplication
    hipEventRecord(startTime, 0);
    kernelMatrixMul << <threadsPerBlock, blocksPerGrid >> > (deviceResultMatrix, deviceMatrixA, deviceMatrixB, SIZE);

    // Stop recording and store the time result in time variable
    hipEventRecord(stopTime, 0);
    hipEventSynchronize(stopTime);
    hipEventElapsedTime(&time, startTime, stopTime);

    // Copy device matrix to host
    hipMemcpy(Result, deviceResultMatrix, BYTES, hipMemcpyDeviceToHost);

    // Print results
    printf("GPU Matrix Multiplication Time: %.2f\n", time);

    // Free event and memory
    hipEventDestroy(startTime);
    hipEventDestroy(stopTime);
    hipFree(deviceMatrixA);
    hipFree(deviceMatrixB);
    hipFree(deviceResultMatrix);
    hipDeviceReset();
}

__global__ void kernelMultipleMatrixMul(float* Result, float* MatrixA, float* MatrixB, int SIZE) {
    // Calculate thread row and columns 
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    // Matrix multiplication with threads
    if (Row < SIZE && Col < SIZE)
    {
        float sum = 0;
        for (int k = 0; k < SIZE; ++k)
            sum += MatrixA[Row * SIZE + k] * MatrixB[k * SIZE + Col];
        Result[Row * SIZE + Col] = sum;
    }

    return;
}

void gpuThreadMatrixMul(float* Result, float* MatrixA, float* MatrixB, int SIZE, int blockWidth) {
    // Determine byte size
    int BYTES = SIZE * SIZE * sizeof(float);

    // Initialize and allocate memory to device matrices
    float* deviceMatrixA;
    float* deviceMatrixB;
    float* deviceResultMatrix;

    hipMalloc(&deviceMatrixA, BYTES);
    hipMalloc(&deviceMatrixB, BYTES);
    hipMalloc(&deviceResultMatrix, BYTES);

    // Copy host matrices to the device 
    hipMemcpy(deviceMatrixA, MatrixA, BYTES, hipMemcpyHostToDevice);
    hipMemcpy(deviceMatrixB, MatrixB, BYTES, hipMemcpyHostToDevice);

    // Initialize time and CUDA events
    float time = 0;

    hipEvent_t startTime, stopTime;
    hipEventCreate(&startTime);
    hipEventCreate(&stopTime);
    hipDeviceSynchronize();

    // Determine factors for grid and block dimenstions
    int NumBlocks = SIZE / blockWidth;
    if (SIZE % blockWidth) NumBlocks++;

    dim3 dimGrid(NumBlocks, NumBlocks);
    dim3 dimBlock(blockWidth, blockWidth);
    
    // Start recording and call the thread multiplication
    hipEventRecord(startTime, 0);
    kernelMultipleMatrixMul << <dimGrid, dimBlock >> > (deviceResultMatrix, deviceMatrixA, deviceMatrixB, SIZE);
    // Stop recording and store time in appropriate variable
    hipEventRecord(stopTime, 0);
    hipEventSynchronize(stopTime);
    hipEventElapsedTime(&time, startTime, stopTime); 

    // Copy the finalized matrix over to the host
    hipMemcpy(Result, deviceResultMatrix, BYTES, hipMemcpyDeviceToHost);

    // Print results
    printf("GPU Matrix Multiplication Time for %d size and %d block width: %.2f\n", SIZE, blockWidth, time);

    // Free memory and events
    hipEventDestroy(startTime);
    hipEventDestroy(stopTime);
    hipFree(deviceMatrixA);
    hipFree(deviceMatrixB);
    hipFree(deviceResultMatrix);
    hipDeviceReset();
}

int main()
{
    //// Number of devices
    //int nd;

    //// Name of Devices
    //char name[50];

    //// Print Number of Cuda Devices
    //hipGetDeviceCount(&nd);
    //printf("Number of CUDA devices: %d\n", nd);

    //// Gather device information for each identified CUDA device 
    //for (int d = 0; d < nd; d++)
    //{
    //    // Struct with device information
    //    hipDeviceProp_t dp;
    //    hipGetDeviceProperties(&dp, d);

    //    // Device Type
    //    printf("Device Type: %s\n", dp.name);
    //    //Clock Rate
    //    printf("Clock Rate: %d\n", dp.clockRate);

    //    // Number of streaming multiprocessors
    //    printf("Number of Streaming Multiprocessors: %d\n", dp.multiProcessorCount);

    //    // Number of cores
    //    printf("Number of Cores: %d\n", 128 * dp.multiProcessorCount);

    //    // Warp size
    //    printf("Warp Size: %d\n", dp.warpSize);

    //    // Global, constant, and shared memory
    //    printf("Global Memory: %zu\n", dp.totalGlobalMem);
    //    printf("Amount of Constant Memory: %zu\n", dp.totalConstMem);
    //    printf("Amount of Shared Memory per Block: %zu\n", dp.sharedMemPerBlock);

    //    // Reg and threads per block
    //    printf("Number of Registers Available Per Block: %d\n", dp.regsPerBlock);
    //    printf("Maximum Number of Threads Per Block: %d\n", dp.maxThreadsPerBlock);

    //    // Dimension of block and grid
    //    printf("Maximum Size of each Dimension of a Block: %dx%dx%d\n", dp.maxThreadsDim[0], dp.maxThreadsDim[1], dp.maxThreadsDim[2]);
    //    printf("Maximum Size of each Dimension of a Grid: %dx%dx%d\n", dp.maxGridSize[0], dp.maxGridSize[1], dp.maxGridSize[2]);

    //}

    //transferFunction(256);
    //transferFunction(512);
    //transferFunction(1024);
    //transferFunction(2048);
    //transferFunction(4096);
    
  // Initialize matrices
  /*float* MatrixA;
  float* MatrixB;
  float* resultMatrix1;
  float* resultMatrix2;*/

  // Initialize to 256
  //int BYTES = 0;

  //BYTES = 256 * 256 * sizeof(float);

  //MatrixA = (float*)malloc(BYTES);
  //MatrixB = (float*)malloc(BYTES);
  //resultMatrix1 = (float*)malloc(BYTES);
  //resultMatrix2 = (float*)malloc(BYTES);

  //int sizes[5] = {256, 512, 1024};

  ////For the three required matrix sizes
  //for(int i = 0; i < 5; i++){
  //    BYTES = sizes[i] * sizes[i] * sizeof(float);

  //    // allocated the necessary memory
  //    MatrixA = (float*)realloc(MatrixA, BYTES);
  //    MatrixB = (float*)realloc(MatrixB, BYTES);
  //    resultMatrix1 = (float*)realloc(resultMatrix1, BYTES);
  //    resultMatrix2 = (float*)realloc(resultMatrix2, BYTES);

  //    // Reinitialize larger arrays
  //    initializeMatrix(MatrixA, sizes[i]);
  //    initializeMatrix(MatrixB, sizes[i]);

  //    // Call CPU multiplication
  //    matrixMul(resultMatrix2, MatrixA, MatrixB, sizes[i]);
  //    // Call GPU multiplication
  //    gpuMatrixMul(resultMatrix1, MatrixA, MatrixB, sizes[i]);
  //    // Verify they are the same value
  //    verifyMatrix(resultMatrix1, resultMatrix2, sizes[i]);

  //}

  //return 0;

  // Initialize all matrices
  float* MatrixA;
  float* MatrixB;
  float* resultMatrix1;
  float* resultMatrix2;

  // Set matrices to first memory size
  int BYTES = 0;

  BYTES = 256 * 256 * sizeof(float);

  MatrixA = (float*)malloc(BYTES);
  MatrixB = (float*)malloc(BYTES);
  resultMatrix1 = (float*)malloc(BYTES);
  resultMatrix2 = (float*)malloc(BYTES);

  // Initialize matrices
  initializeMatrix(MatrixA, 256);
  initializeMatrix(MatrixB, 256);

  // Thread block sizes/ matrix sizes
  int list[5] = { 2, 4, 8, 16, 32 };
  int sizes[5] = {256, 512, 1024, 2048, 4096};

  // Loop through sizes
  for(int i = 0; i < 5; i++){
      BYTES = sizes[i] * sizes[i] * sizeof(float);

      // Allocate memory for new size
      MatrixA = (float*)realloc(MatrixA, BYTES);
      MatrixB = (float*)realloc(MatrixB, BYTES);
      resultMatrix1 = (float*)realloc(resultMatrix1, BYTES);
      resultMatrix2 = (float*)realloc(resultMatrix2, BYTES);

      // Initialize larger matrices
      initializeMatrix(MatrixA, sizes[i]);
      initializeMatrix(MatrixB, sizes[i]);

      for (int j = 0; j < 5; j++) {
          // Call GPU matrix multiplications with differeing threads
          gpuThreadMatrixMul(resultMatrix1, MatrixA, MatrixB, sizes[i], list[j]);
          // Call CPU matrix multiplication
          matrixMul(resultMatrix2, MatrixA, MatrixB, sizes[i]);
          // Ensure the multiplcation is the same result
          verifyMatrix(resultMatrix1, resultMatrix2, sizes[i]);
      }
  }

  return 0;
}


