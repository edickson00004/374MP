#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <ctime>

void transferFunction(int SIZE) {
    int BYTES = SIZE * SIZE * sizeof(float);

    float* hostMatrixA = 0;
    float* hostMatrixB = 0;
    hipHostMalloc((void**)&hostMatrixA, BYTES);
    hipHostMalloc((void**)&hostMatrixB, BYTES);

    float* deviceMatrixA = 0;
    float* deviceMatrixB = 0;

    hipMalloc((void**)&deviceMatrixA, BYTES);
    hipMalloc((void**)&deviceMatrixB, BYTES);

    float HTD = 0;
    float DTH = 0;

    hipEvent_t startTime, stopTime;
    hipEventCreate(&startTime);
    hipEventCreate(&stopTime);
    hipDeviceSynchronize();

    hipEventRecord(startTime, 0);
    hipMemcpy(deviceMatrixA, hostMatrixA, BYTES, hipMemcpyHostToDevice);
    hipMemcpy(deviceMatrixB, hostMatrixB, BYTES, hipMemcpyHostToDevice);
    hipEventRecord(stopTime); // stop is updated here
    hipEventSynchronize(stopTime);
    hipEventElapsedTime(&HTD, startTime, stopTime); //time difference between start and stop

    hipEventRecord(startTime, 0);
    hipMemcpy(hostMatrixA, deviceMatrixA, BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(hostMatrixB, deviceMatrixB, BYTES, hipMemcpyDeviceToHost);
    hipEventRecord(stopTime); // stop is updated here
    hipEventSynchronize(stopTime);
    hipEventElapsedTime(&DTH, startTime, stopTime);


    printf("Host to Device: % .2fms\n", HTD);
    printf("Device to Host: %.2fms\n", DTH);

    hipEventDestroy(startTime);
    hipEventDestroy(stopTime);
    hipHostFree(hostMatrixA);
    hipHostFree(hostMatrixB);
    hipFree(deviceMatrixA);
    hipFree(deviceMatrixB);
    hipDeviceReset();
}

void initializeMatrix(float* Array, int SIZE) {
    for (int i = 0; i < SIZE; i++) {
        for (int j = 0; j < SIZE; j++) {
            Array[i * SIZE + j] = (float)(rand() / RAND_MAX);

        }
    }

}
void matrixMul(int SIZE) {
    int BYTES = SIZE * SIZE * sizeof(float);

    float* MatrixA = (float*)malloc(BYTES);
    float* MatrixB = (float*)malloc(BYTES);
    float* resultMatrix = (float*)malloc(BYTES);

    for (int i = 0; i < SIZE; i++) {
        for (int j = 0; j < SIZE; j++) {
            MatrixA[i * SIZE + j] = 1;
            MatrixB[i * SIZE + j] = 1;

        }
    }
    clock_t startTime, stopTime;

    startTime = clock();

    for (int i = 0; i < SIZE; i++) {
        for (int j = 0; j < SIZE; j++) {
            resultMatrix[i * SIZE + j] = 0;

            for (int k = 0; k < SIZE; k++) {
                resultMatrix[i * SIZE + j] += MatrixA[i * SIZE + k] * MatrixB[k * SIZE + j];
            }
        }
    }

    stopTime = clock();

    printf("CPU Matrix Multiplication % .2fms\n", (double)stopTime - startTime);

    free(MatrixA);
    free(MatrixB);
    free(resultMatrix);

}
__global__ void kernelMatrixMul(float* Result, float* MatrixA, float* MatrixB, int SIZE) {

    for (int i = 0; i < SIZE; i++) {
        for (int j = 0; j < SIZE; j++) {
            Result[i * SIZE + j] = 0;
            for (int k = 0; k < SIZE; k++) {
                Result[i * SIZE + j] += MatrixA[i * SIZE + k] * MatrixB[k * SIZE + j];
            }
        }
    }


    return;
}
void gpuMatrixMul(float* Result, float* MatrixA, float* MatrixB, int SIZE) {
    int BYTES = SIZE * SIZE * sizeof(float);

    float* deviceMatrixA;
    float* deviceMatrixB;
    float* deviceResultMatrix;

    hipMalloc(&deviceMatrixA, BYTES);
    hipMalloc(&deviceMatrixB, BYTES);
    hipMalloc(&deviceResultMatrix, BYTES);

    hipMemcpy(deviceMatrixA, MatrixA, BYTES, hipMemcpyHostToDevice);
    hipMemcpy(deviceMatrixB, MatrixB, BYTES, hipMemcpyHostToDevice);

    float time = 0;

    hipEvent_t startTime, stopTime;
    hipEventCreate(&startTime);
    hipEventCreate(&stopTime);
    hipDeviceSynchronize();

    dim3 threadsPerBlock(1, 1);
    dim3 blocksPerGrid(1, 1);

    hipEventRecord(startTime, 0);
    kernelMatrixMul << <threadsPerBlock, blocksPerGrid >> > (deviceResultMatrix, deviceMatrixA, deviceMatrixB, SIZE);
    hipEventRecord(stopTime, 0); // stop is updated here
    hipEventSynchronize(stopTime);
    hipEventElapsedTime(&time, startTime, stopTime); //time difference between start and stop

    hipMemcpy(Result, deviceResultMatrix, BYTES, hipMemcpyDeviceToHost);

    printf("GPU Matrix Multiplication Time: %.2f", time);
    hipEventDestroy(startTime);
    hipEventDestroy(stopTime);
    hipFree(deviceMatrixA);
    hipFree(deviceMatrixB);
    hipFree(deviceResultMatrix);
    hipDeviceReset();
}

int main()
{
    int nd;
    char name[50];

    hipGetDeviceCount(&nd);

    printf("Number of CUDA devices: %d\n", nd);
    for (int d = 0; d < nd; d++)
    {
        hipDeviceProp_t dp;
        hipGetDeviceProperties(&dp, d);
        printf("Device Type: %s\n", dp.name);
        printf("Clock Rate: %d\n", dp.clockRate);
        printf("Number of Streaming Multiprocessors: %d\n", dp.multiProcessorCount);
        printf("Number of Cores: %d\n", 128 * dp.multiProcessorCount);
        printf("Warp Size: %d\n", dp.warpSize);
        printf("Global Memory: %zu\n", dp.totalGlobalMem);
        printf("Amount of Constant Memory: %zu\n", dp.totalConstMem);
        printf("Amount of Shared Memory per Block: %zu\n", dp.sharedMemPerBlock);
        printf("Number of Registers Available Per Block: %d\n", dp.regsPerBlock);
        printf("Maximum Number of Threads Per Block: %d\n", dp.maxThreadsPerBlock);
        printf("Maximum Size of each Dimension of a Block: %dx%dx%d\n", dp.maxThreadsDim[0], dp.maxThreadsDim[1], dp.maxThreadsDim[2]);
        printf("Maximum Size of each Dimension of a Grid: %dx%dx%d\n", dp.maxGridSize[0], dp.maxGridSize[1], dp.maxGridSize[2]);

    }

    transferFunction(256);
    transferFunction(512);
    transferFunction(1024);
    transferFunction(2048);
    transferFunction(4096);

    matrixMul(256);
    matrixMul(512);
    matrixMul(1024);

    float* MatrixA = 0;
    float* MatrixB = 0;
    float* resultMatrix = 0;
    int BYTES = 0;

    BYTES = 256 * 256 * sizeof(float);

    MatrixA = (float*)malloc(BYTES);
    MatrixB = (float*)malloc(BYTES);
    resultMatrix = (float*)malloc(BYTES);
    initializeMatrix(MatrixA, 256);
    initializeMatrix(MatrixB, 256);

    gpuMatrixMul(resultMatrix, MatrixA, MatrixB, 256);

    return 0;
}

