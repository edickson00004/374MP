#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <string.h>
#include <stdio.h>

/*
__global__ void GPU_matrix_multiply(float* M, float* N, float* P, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float temp = 0;
    for (int i = 0; i < n; i++) {
     temp += M[row * n + i] * N[i * n + col];
    }
    P[row * n + col] = temp;
}
*/
__global__ void GPU_matrix_multiply(float* M, float* N, float* P, int n) {

    for (int i = 0; i < n; i++) {
        float temp = 0;
        for (int j = 0; j < n; j++) {
            for (int k = 0; k < n; k++) {
                temp += M[i * n + k] * N[k * n + j];
            }
            P[i * n + j] = temp;
        }
    }
}


void gpu_Multi(float* M, float* N, float* P, int n) {
    size_t size = n * n * sizeof(float);
    float* d_M;
    float* d_N;
    float* d_P;
    hipMalloc((void**)&d_P, size);
    hipMalloc((void**)&d_M, size);
    hipMalloc((void**)&d_N, size);

    hipMemcpy(d_M, M, size, hipMemcpyHostToDevice);
    hipMemcpy(d_N, N, size, hipMemcpyHostToDevice);
    float time = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipDeviceSynchronize();
    
    dim3 threads(1, 1);
    dim3 blocks(1, 1);
    hipEventRecord(start, 0);
    GPU_matrix_multiply << < blocks, threads >> > (d_M, d_N, d_P, n);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipMemcpy(P, d_P, size, hipMemcpyDeviceToHost);
    printf("GPU time: %f\n", time);

}

void cpu_Mulit(float* M, float* N, float* P, int n) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipDeviceSynchronize();
    hipEventRecord(start, 0);
    float time;
    for (int i = 0; i < n; i++) {
        float temp = 0;
        for (int j = 0; j < n; j++) {
            for (int k = 0; k < n; k++) {
                temp += M[i * n + k] * N[k * n + j];
            }
            P[i * n + j] = temp;
        }
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("CPU time: %f\n", time);
}

void transferTime(float* M, float* N, int size) {
    float* d_M;
    float* d_N;
    float deviceTime, hostTime;
    hipMalloc((void**)&d_M, size);
    hipMalloc((void**)&d_N, size);
    hipEvent_t start, stop;

    //From host to device
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipDeviceSynchronize();
    hipEventRecord(start, 0);
    hipMemcpyAsync(d_M, M, size, hipMemcpyHostToDevice, 0);
    hipMemcpyAsync(d_N, N, size, hipMemcpyHostToDevice, 0);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&deviceTime, start, stop);

    //from device to host
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipDeviceSynchronize();
    hipEventRecord(start, 0);
    hipMemcpyAsync(M, d_M, size, hipMemcpyDeviceToHost, 0);
    hipMemcpyAsync(N, d_N, size, hipMemcpyDeviceToHost, 0);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&hostTime, start, stop);

    printf("Host to Device time: %f\n", deviceTime);
    printf("Device to Host time: %f\n", hostTime);

}

void genRandMatrix(float* matrix, int n) {
    int limit = n * n;
    for (int i = 0; i < limit; i++) {
        matrix[i] = (float)rand() / RAND_MAX;
    }
}
int main()
{
    int n = 256;
    size_t size = n * n * sizeof(float);
    float* M = (float*)malloc(size);
    float* N = (float*)malloc(size);
    float* P = (float*)malloc(size);

    genRandMatrix(M, n);
    genRandMatrix(N, n);

    /* Part 2.1
        for (int i = 0; i < 10; i++) {
        printf("Trial %d -----------------------------------\n", i + 1);
        genRandMatrix(M, n);
        genRandMatrix(N, n);
        transferTime(M, N, size);
    }
    */
    //Part 2.2
    gpu_Multi(M, N, P, n);
    cpu_Mulit(M, N, P, n);



    return 0;
}